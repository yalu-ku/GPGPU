
#include <hip/hip_runtime.h>
#include <stdio.h>

void verify1D(float * N, float * P, float * mask, int width, int mask_width);

static void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define WIDTH 10000 //Input Vector 길이
#define MASK_WIDTH 5
#define O_TILE_WIDTH 1020
#define BLOCK_WIDTH (O_TILE_WIDTH + 4)

__global__ void Conv1D(float * N, float * P, float * Mask, int width, int mask_width)
{
    float output = 0.0f;
    int tx = threadIdx.x;
    int index_o = blockIdx.x * O_TILE_WIDTH + threadIdx.x;
    int index_i = index_o - 2;
    __shared__ float Ns[BLOCK_WIDTH];

    if((index_i>=0) && (index_i < width)) {
        Ns[tx] = N[index_i];
    } else {
        Ns[tx] = 0.0f;
    }
    __syncthreads();

    if((threadIdx.x < O_TILE_WIDTH) && (index_o < width)) {
        output = 0.0f;
        for(int j=0; j < mask_width; j++) {
            output += Mask[j] * Ns[j+threadIdx.x];
        }
        P[index_o] = output;
    }
}

int main()
{
    float *N, *P, *Mask;
    float *dev_N, *dev_P, *dev_Mask;

    N = (float*)malloc(sizeof(float)*WIDTH);
    P = (float*)malloc(sizeof(float)*WIDTH);
    Mask = (float*)malloc(sizeof(float)*5);

    // Initialize
    for (int i=0; i<WIDTH; i++) {
        N[i] = (rand()%100)/100.00;
    }
    for (int i=0; i<5; i++) {
        Mask[i] = (rand()%100)/100.00;
    }

    // Add vectors in parrallel
    HANDLE_ERROR(hipMalloc((void**)&dev_N, WIDTH * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_P, WIDTH * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_Mask, WIDTH * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(dev_N, N, WIDTH * sizeof(float), hipMemcpyHostToDevice));    HANDLE_ERROR(hipMemcpy(dev_Mask, Mask, WIDTH * sizeof(float), hipMemcpyHostToDevice));

    // Launch a kernel on the GPU with one thread for each element.
    dim3 dimBlock(BLOCK_WIDTH, 1, 1);
    dim3 dimGrid((WIDTH-1)/O_TILE_WIDTH+1, 1, 1);
    Conv1D<<<dimGrid, dimBlock>>>(dev_N, dev_P, dev_Mask, WIDTH, MASK_WIDTH);
    HANDLE_ERROR(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    HANDLE_ERROR(hipMemcpy(P, dev_P, WIDTH * sizeof(float), hipMemcpyDeviceToHost));

    verify1D(N, P, Mask, WIDTH, MASK_WIDTH);

    hipFree(dev_N);
    hipFree(dev_P);
    hipFree(dev_Mask);
    free(N);
    free(P);
    free(Mask);
    return 0;
}

void verify1D(float * N, float * P, float * mask, int width, int mask_width)
{
    const float relativeTolerance = 1e-6;
    for(int i=0; i<width; ++i) {
        float sum = 0.0f;
        for(int i_m=0; i_m<mask_width; ++i_m) {
            int iN = i + i_m - (int)(mask_width/2);
            if(iN>=0 && iN<width) {
                sum += mask[i_m]*N[iN];
            }
        }
        float relativeError = (sum - P[i])/sum;
        if (relativeError > relativeTolerance || relativeError < -relativeTolerance) {
            printf("TEST FAILED\n\n");
            exit(0);
        }
    }
    printf("TEST PASSED\n\n");
}