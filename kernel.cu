﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


__global__ void kernel(int* a, int dimx, int dimy)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * dimx + ix;

    a[idx] = a[idx] + 1;
}

int main()
{
    int dimx = 16;
    int dimy = 16;
    int num_bytes = dimx * dimy * sizeof(int);

    int* d_a = 0, * h_a = 0; // device and host pointers

    h_a = (int*)malloc(num_bytes);
    hipMalloc((void**)&d_a, num_bytes);

    if (0 == h_a || 0 == d_a)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset(d_a, 0, num_bytes);

    dim3 grid, block;
    block.x = 4;
    block.y = 4;

    // grid???
    //grid.x = 0;
    //grid.y = 0;
    grid.x = dimx / block.x;
    grid.y = dimy / block.y;

    kernel <<< grid, block >>> (d_a, dimx, dimy);

    // cudaMemcpy
    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

    for (int row = 0; row < dimy; row++)
    {
        for (int col = 0; col < dimx; col++)
            printf("%d ", h_a[row * dimx + col]);
        printf("\n");
    }

    free(h_a);
    hipFree(d_a);

    return 0;
}
